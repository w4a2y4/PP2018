#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define b 4

__global__ void hello(int A[Size][Size], int B[Size][Size], int C[Size][Size])
{
	/*Your kernel*/
}

void metric_mul(int A[Size][Size], int B[Size][Size], int C[Size][Size])
{
	/*Your initail*/
    int *device_A, *device_B, *device_C;
    dim3 blocks(b, b);
    dim3 grids(Size/b,Size/b);
    int size = sizeof(int ) * Size * Size;
    hipMalloc((void **)&device_A, size);
    hipMalloc((void **)&device_B, size);
    hipMalloc((void **)&device_C, size);

    hipMemcpy(device_A, (int *)A, sizeof(int) * Size * Size, hipMemcpyHostToDevice);
    hipMemcpy(device_B, (int *)B, sizeof(int) * Size * Size, hipMemcpyHostToDevice);
    hello <<< grids, blocks>>> ((int (*)[Size])device_A, (int (*)[Size])device_B, (int (*)[Size])device_C);

    hipMemcpy((int *)C, device_C, size, hipMemcpyDeviceToHost);
    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);
    return ;
}
