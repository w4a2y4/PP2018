#include <stdio.h>
#include <hip/hip_runtime.h>  

#ifndef Size
#define Size 20
#endif

__global__ void hello(int A[Size][Size], int B[Size][Size], int C[Size][Size])  
{  
    int k, sum = 0;
    for (k = 0; k < Size; k++)
	sum += A[threadIdx.x][k] * B[k][threadIdx.y];

    C[threadIdx.x][threadIdx.y] = sum;
}  
  
int main(void)  
{  
    int *device_A, *device_B, *device_C;
    int *host_A, *host_B, *host_C;
    int i, j, k;
    int size = sizeof(int) * Size * Size;
    int *aptr, *bptr;
    
    dim3 blocks(Size, Size);

    hipMalloc((void **)&device_A, size);
    hipMalloc((void **)&device_B, size);
    hipMalloc((void **)&device_C, size);
    host_A = (int *)malloc(size);
    host_B = (int *)malloc(size);
    host_C = (int *)malloc(size);

    aptr = host_A;
    bptr = host_B;
    for (i = 0; i < Size; i++)
     	for (j = 0; j < Size; j++) {
	    *aptr++ = i + j;
	    *bptr++ = i + j;
	}
	    
    hipMemcpy(device_A, host_A, size, hipMemcpyHostToDevice);  
    hipMemcpy(device_B, host_B, size, hipMemcpyHostToDevice);  

    hello <<< 1, blocks >>> ((int (*)[Size])device_A, (int (*)[Size])device_B, 
			     (int (*)[Size])device_C);  
    hipMemcpy(host_C, device_C, size, hipMemcpyDeviceToHost);

    k = 0;
    for (i = 0; i < Size; i++)
	for (j = 0; j < Size; j++)
	    printf("host_C[%d][%d] = %d\n", i, j, host_C[k++]);

    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);
    free(host_A);
    free(host_B);
    free(host_C);
}
