#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#ifndef Size
#define Size 1000

#endif
#define b 4

void metric_mul_gold(int A[Size][Size], int B[Size][Size], int C[Size][Size])
{

    int i,j,k;
	for(i=0; i<Size; i++)
		for(j=0; j<Size; j++)
			for(k=0; k<Size; k++)
				C[i][j] += A[i][k]*B[k][j];
    return;
}


void metric_mul(int A[Size][Size], int B[Size][Size], int C[Size][Size]);

int main(void)
{
    int i, j, k;
    int size = sizeof(int) * Size * Size;
    int *aptr, *bptr, *cptr;

    int *host_A, *host_B, *host_C;

    srand(time(NULL));

    host_A = (int *)malloc(size);
    host_B = (int *)malloc(size);
    host_C = (int *)malloc(size);

    aptr = host_A;
    bptr = host_B;
	cptr = host_C;
    for (i = 0; i < Size; i++)
        for (j = 0; j < Size; j++) {
            *aptr++ = rand() % 10;
            *bptr++ = rand() % 10;
			*cptr++ = 0;
        }
		
	int *gold_C;
    gold_C = (int *)malloc(size);
    metric_mul_gold((int (*)[Size])host_A, (int (*)[Size])host_B, (int (*)[Size])gold_C);
	
    hipEvent_t start_time, stop_time;
    float exectime;
    hipEventCreate(&start_time);
    hipEventCreate(&stop_time);
    hipEventRecord(start_time, 0);
    metric_mul((int (*)[Size])host_A, (int (*)[Size])host_B, (int (*)[Size])host_C);
    hipEventRecord(stop_time, 0);
    hipEventSynchronize(stop_time);
    hipEventElapsedTime(&exectime, start_time, stop_time);
    
    printf("real %f ms\n ", exectime);
    hipEventDestroy(start_time);
    hipEventDestroy(stop_time);

    //check result
    if (!memcmp(host_C, gold_C, size))
    	printf("AC!\n");
    else
    	printf("Failed!\n");
    	
    /*k = 0;
    for (i = 0; i < Size; i++)
        for (j = 0; j < Size; j++)
            printf("host_C[%d][%d] = %d\n", i, j, host_C[k++]);*/

    free(host_A);
    free(host_B);
    free(host_C);
    free(gold_C);
    return 0;
}
